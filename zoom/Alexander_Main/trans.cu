#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <iostream>
#include <time.h>

using namespace std;

#define BLOCK_COUNT 64
#define THREAD_COUNT 1024

__global__ void muliply(unsigned int *a, unsigned int *b, unsigned int *c, unsigned int rowA, unsigned int colA, unsigned int rowB, unsigned int colB)
{
    unsigned int tileID = blockIdx.x;
    unsigned int outputMatrix[65536]; //256*256 size.
    unsigned int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int smem[THREAD_COUNT];
    unsigned int startingRow;
    unsigned int startingCol;
    unsigned int lastRow;
    unsigned int lastCol;
    unsigned int shm_row = 0;
    unsigned int shm_col = 0;
    

    //transpose block to shared memory
    startingRow = (tileID / 8) * 32;//this is the starting row in the matrix tile transpose.
    startingCol = (tileID % 8) * 32;//this is the starting Col in the matrix tile to transpose.
    lastRow = startingRow + 32;
    lastCol = startingCol + 32;
    

    smem[shm_row * shm_col] = 


}

__global__ void multiply2(unsigned int *a, unsigned int *b, unsigned int *c, unsigned int rowA, unsigned int colA, unsigned int rowB, unsigned int colB)
{
    __shared__ int smem[THREAD_COUNT];
    unsigned int currentRow = blockIdx.x;
    while(currentRow < rowA)
    {
        for(int cB = 0; cB < colB; cB++)
        {
            int tid = threadIdx.x;
            int res = 0;
            while(threadIdx.x < colA)
            {
                res + = a[currentRow* blockDim.x + tid] * b[tid * colB + cB];
                tid += blockDim.x;
            }
            smem[threadIdx.x] = res; //this needs a lot of work! This needs to go above!
            __syncthreads();

            for(unsigned int i = blockDim.x/2; i > 0; i = i/2)
            {
                if(threadIdx.x < i)
                {
                    smem[threadIdx.x] = smem[threadIdx.x] + smem[threasIdx.x + i];
                }
                __syncthreads();
            }
            if(threadIdx.x == 0){c[colB * currentRow + cB] = smem[0];
        }
        currentRow += blockDim.x;
    }
}

int main(void)
{

	unsigned int *A, *B, *C;
	unsigned int B_Col = 256;
	unsigned int A_Col = 256;
	unsigned int B_Row = 256;
	unsigned int A_Row = 256;
	unsigned int C_Size = A_Row * B_Col;
    unsigned int B_Size = B_Row * B_Col;
    unsigned int A_Size = A_Row * A_Col;
	clock_t t1, t2;

	t1 = clock();

	A = new(nothrow) unsigned int[A_Size];
	B = new(nothrow) unsigned int[B_Size];
	C = new(nothrow) unsigned int[C_Size];

	for(unsigned int i = 0; i < A_Size; i++){A[i] = i % 10;}
    for(unsigned int i = 0; i < B_Size; i++){B[i] = i % 10;}

	unsigned int *A_D, *B_D, *C_D;

	//Dynamic Memory Allocation
	hipMalloc((void**)&A_D, sizeof(unsigned int) * A_Size);
	hipMalloc((void**)&B_D, sizeof(unsigned int) * B_Size);
	hipMalloc((void**)&C_D, sizeof(unsigned int) * C_Size);

	//copy the contents of matric C to corressondingly allocated matrix on the GPU
	hipMemcpy(A_D, A, sizeof(unsigned int) * A_Size, hipMemcpyHostToDevice);
	hipMemcpy(B_D, B, sizeof(unsigned int) * B_Size, hipMemcpyHostToDevice);

	//Call GPU Kernel.
	multiply<<<BLOCK_COUNT, THREAD_COUNT>>>(A_D, B_D, C_D, A_Row, A_Col, B_Row, B_Col);
	//copy the values of the allocated GPU C_D array to the C array on the CPU.
	hipMemcpy(C, C_D, sizeof(unsigned int) * C_Size, hipMemcpyDeviceToHost);

	for(unsigned int i = 0; i < C_Size; i++){cout << " " << C[i];}	
	
	//calculate the time difference.
	t2 = clock();
	double difference = ((double)t2-(double)t1);
    double seconds = difference / CLOCKS_PER_SEC;

	//cout << endl << "Run time: " << seconds << " seconds" << endl;
	cout << ;
	hipFree(A_D); hipFree(B_D); hipFree(C_D); //Free dynamically allocated memory

	return 0;
}	
