#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__global__ void gpu_matrix_vec_mult(int *A,int *b, int *C, int m, int n)
{ 
    
    __shared__ int smem[256];
    int myrow = blockIdx.x;
    while(myrow<m)
    {
    	smem[threadIdx.x] = A[myrow*n+threadIdx.x];
    	smem[threadIdx.x] *= b[threadIdx.x];
    	//parallel reduction
    	__syncthreads();
    	for ( int i = blockDim.x/2; i > 0; i = i /2)
    	{
    		if (threadIdx.x < i)
    		{
    			int temp = smem[ threadIdx.x] + smem[threadIdx.x + i];
    			smem[threadIdx.x] = temp; 
    		}
    		__syncthreads();
    	}
    	if(threadIdx.x==0)
    		C[myrow] = smem[0];
    	//after 128 blocks execution, "move" to next 128 rows for 128 blocks
    	myrow+=128;
    }
}

void cpu_matrix_vec_mult(int *A, int *b, int *result, int m, int n) {
    for (int i = 0; i < m; ++i) 
    {
        int tmp = 0;
        for (int h = 0; h < n; ++h) 
        {
        	tmp += A[i * n + h] * b[h];
        }
        result[i] = tmp;
    }
}

int main(int argc, char const *argv[])
{
    int m, n;
    /* seed for illustration */
    srand(3333);
    printf("please enter m n\n");
    scanf("%d %d", &m, &n);

    // allocate memory in host RAM, h_c is used to store CPU result
    int *A, *b, *C, *h_c;
    hipHostMalloc((void **) &A, sizeof(int)*m*n);
    hipHostMalloc((void **) &b, sizeof(int)*n*1);
    hipHostMalloc((void **) &C, sizeof(int)*m*1);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*1);

    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            A[i * n + j] = rand() % 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        b[i] = rand() % 1024;
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    // Allocate memory space on the device 
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*1);
    hipMalloc((void **) &d_c, sizeof(int)*m*1);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, A, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int)*n*1, hipMemcpyHostToDevice);

    // Launch kernel
    gpu_matrix_vec_mult<<<128, 256>>>(d_a, d_b, d_c, m, n);  
    // Transefr results from device to host 
    hipMemcpy(C, d_c, sizeof(int)*m*1, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    // printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);
    printf("GPU time use on matrix multiplication is : %f ms\n", gpu_elapsed_time_ms);

    // start the CPU version
    hipEventRecord(start, 0);
    cpu_matrix_vec_mult(A, b, h_c, m, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("CPU time use on matrix multiplication is : %f ms",cpu_elapsed_time_ms);


    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
            //printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_c[i*k + j], i, j, h_c[i*k + j]);
        if(C[i] != h_c[i])
        {
            all_ok = 0;
        }
    }

    // roughly compute speedup
    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(A);
    hipHostFree(b);
    hipHostFree(C);
    return 0;
}






