#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// #define num_size 16000000



__global__ void gpu_add_atom(int *a, int *res, int num_size)
{

	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // while(idx < num_size)
    // {
    // 	for(int i = 0; i < 128 * 128; i++)
    // 	{
    // 		atomicAdd(res, a[i]);
    // 	}
    // 	idx += 128*128;

    // }

    for (unsigned int i = idx; i < num_size; i += blockDim.x*gridDim.x)
    {
        atomicAdd(res, a[i]);
    }

}



	// int i = atomicAdd(index,1);

	// int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// sum[i] = idx;

//CPU version. Mainly for validation
void cpu_add(int *a, int *result, int num_size)
{
    for (int i = 0; i < num_size; i++) 
    {
    	result[0] += a[i];

    }
}


int main()
{
	int *a;
	int *res;
	int *h_res;
	int num_size = 160000000; 
	hipHostMalloc((void **) &a, sizeof(int)*num_size);
	hipHostMalloc((void **) &res, sizeof(int)*1);
	hipHostMalloc((void **) &h_res, sizeof(int)*1);

    for (int i = 0; i < num_size; i++) {
        a[i] = rand() % 5;
    }

    printf("a[0] in main: %d\n", a[0]);
    res[0] = 0;
    h_res[0] = 0;


    float gpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);

    int *d_a;
    int *d_res;
    hipMalloc((void **) &d_a, sizeof(int)*num_size);
    hipMalloc((void **) &d_res, sizeof(int)*1);

    hipMemcpy(d_a, a, sizeof(int)*num_size, hipMemcpyHostToDevice);

    gpu_add_atom<<<128, 128>>>(d_a, d_res, num_size);

    hipMemcpy(res, d_res, sizeof(int)*1, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    // printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);
    printf("GPU time use on atomicAdd is : %f ms\n", gpu_elapsed_time_ms);

    cpu_add(a, h_res, num_size);

    printf("gpu result: %d\n", res[0]);
    printf("cpu result: %d\n", h_res[0]);

    if(res[0] == h_res[0]) printf("correct result!\n");

    hipFree(d_a);
    hipFree(d_res);
    hipHostFree(a);
    hipHostFree(res);
    hipHostFree(h_res);
    return 0;


}
